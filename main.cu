#include "hip/hip_runtime.h"
#include "file.h"
#include "hough.h"
#include "laplace.h"

int main(int argc, char ** argv)
{
	int n=1000;
	char inputFile[] = { "image.pgm" };
	char outputFile[] = { "treshold.pgm" };
	char resultFile[] = { "Houghed.pgm" };
	Image *image=readImageFromFile(inputFile);
	/*char testFile[] = { "test.pgm" };
	saveImageToFileTest(testFile);*/
	Image *devImage;
	Image *devThresholdImage;
	Image *result = new Image(image->getWidth(),image->getHeight(),image->getScale());
	double *ro;
	double *theta;
	size_t steps = n*sizeof(int);
	int *A;
	int size=image->getHeight()*image->getWidth();
	size_t size2 = size*sizeof(int);
	int *indexes;
	int *indexesHost;
	int *deviceImage;
	int *B = new int[size];
	hipMalloc((void**)&devImage,sizeof(Image) );
	hipMalloc((void**)&deviceImage,size2 );
	hipMalloc((void**)&indexes,size2);
	hipMalloc((void**)&devThresholdImage,sizeof(Image) );
	hipMalloc((void**)&A,size2 );
	hipMalloc((void**)&ro,sizeof(double)* steps);
	hipMalloc((void**)&theta,sizeof(double)* steps);
	hipMemcpy(devImage,image,sizeof(Image),hipMemcpyHostToDevice);
	hipMemcpy(devThresholdImage,result,sizeof(Image),hipMemcpyHostToDevice);
	printf("End of allocation\n");
	makeLaplaceMask<<<24,7>>>(devImage,
	 devThresholdImage, size);
	//thresholdImage<<<24,7>>>(devThresholdImage,devThresholdImage->getScale()/2,size);
	hipMemcpy(result,devThresholdImage,sizeof(Image), hipMemcpyDeviceToHost);
	hipMemcpy(B,deviceImage,size2, hipMemcpyDeviceToHost);
	for(int i=0;i<size;i++)
	{
		printf("%i ",B[i]);
	}
	printf("After treshold\n");
    saveImageToFile(result,outputFile);
	double roMax = sqrt(image->getHeight()*image->getHeight() + 
	image->getWidth()*image->getWidth());
	createRoAndThetaArrays<<<24,7>>>(ro, theta, roMax/steps, 3.14/steps, steps);
	printf("After Ro theta arrays\n");
	houghTransform<<<24,7>>>(devThresholdImage,
	 ro, theta, A, steps, steps);
	printf("Hough\n");
	findLocalMaximas<<<24,7>>>(A, 5, indexes);
	printf("Local Maximas\n");
	//hipMemcpy(result,devImage,sizeof(Image), hipMemcpyDeviceToHost);
	hipMemcpy(indexesHost,indexes,sizeof(int)
		*image->getHeight()*image->getWidth(),
		hipMemcpyDeviceToHost);
	result->setArray(indexesHost);
	printf("After set array\n");
	saveImageToFile(result,resultFile);
	hipFree(theta);
	hipFree(ro);
	hipFree(A);
	hipFree(devImage);
	hipFree(deviceImage);
	hipFree(indexes);
	hipFree(devThresholdImage);
	free(image);
	free(result);
	free(B);
	return 0;
}
