#include "file.h"
#include "hough.h"
#include "laplace.h"

int main(int argc, char ** argv)
{
	int n=1000;
	char inputFile[] = { "image.pgm" };
	char outputFile[] = { "treshold.pgm" };
	Image *image=readImageFromFile(inputFile);
	char testFile[] = { "test.pgm" };
	saveImageToFileTest(testFile);
	Image *devImage;
	Image *devThresholdImage;
	Image *result = new Image(image->getWidth(),image->getHeight(),image->getScale());
	double *ro;
	double *theta;
	size_t steps = n*sizeof(int);
	int *A;
	int size=image->getHeight()*image->getWidth();
	size_t size2 = size*sizeof(int);
	int *indexes;
	int *indexesHost;
	int *deviceImage;
	int *B = new int[size];
	hipMalloc((void**)&devImage,sizeof(Image) );
	hipMalloc((void**)&deviceImage,size2 );
	hipMalloc((void**)&indexes,size2);
	hipMalloc((void**)&devThresholdImage,sizeof(Image) );
	hipMalloc((void**)&A,size2 );
	hipMalloc((void**)&ro,sizeof(double)* steps);
	hipMalloc((void**)&theta,sizeof(double)* steps);
	hipMemcpy(devImage,image,sizeof(Image),hipMemcpyHostToDevice);
	hipMemcpy(devThresholdImage,image,sizeof(Image),hipMemcpyHostToDevice);
	printf("End of allocation\n");
	makeLaplaceMask<<<24,7>>>(devImage,
	 devThresholdImage, size);
	//thresholdImage<<<24,7>>>(devThresholdImage,devThresholdImage->getScale()/2,size);
	hipMemcpy(result,devThresholdImage,sizeof(Image), hipMemcpyDeviceToHost);
	hipMemcpy(B,deviceImage,size2, hipMemcpyDeviceToHost);
	for(int i=0;i<size;i++)
	{
		printf("%i ",B[i]);
	}
	printf("After treshold\n");
        saveImageToFile(result,outputFile);
	/*createRoAndThetaArrays<<<10,10>>>(ro, theta, 3.14/steps, 3.14/steps, steps);
	printf("After Ro theta arrays\n");
	houghTransform<<<10,10>>>(devThresholdImage,
	 ro, theta, A, 100, 100);
	printf("Hough\n");
	findLocalMaximas<<<10,10>>>(A, 5, indexes);
	printf("Local Maximas\n");
	hipMemcpy(result,devImage,sizeof(Image), hipMemcpyDeviceToHost);
	hipMemcpy(indexesHost,indexes,sizeof(int)
		*image->getHeight()*image->getWidth(),
		hipMemcpyDeviceToHost);
	result->setArray(indexesHost);
	printf("After set array\n");
	saveImageToFile(result,"Houghed.pgm");*/
	hipFree(theta);
	hipFree(ro);
	hipFree(A);
	hipFree(devImage);
	hipFree(deviceImage);
	hipFree(indexes);
	hipFree(devThresholdImage);
	free(image);
	free(result);
	free(B);
	return 0;
}
