#include "hip/hip_runtime.h"
#include "file.h"
#include "hough.h"
#include "laplace.h"

int main(int argc, char ** argv)
{
<<<<<<< HEAD
	char inputFile[] = { "mountain.pgm" };
	char resultFile[] = { "Houghed.pgm" };
	Image *image=readImageFromFile(inputFile);
	int *devImage;
	int *devTresholded;
	int blocks = image->getWidth();
	int threads = image->getHeight();
	int size=image->getHeight()*image->getWidth();
	size_t imageSize = (size+3) * sizeof(int);
	hipMalloc((void**)&devImage,size*sizeof(int) );
	hipMemcpy(devImage,image->getArray(),size*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc((void**)&devTresholded,size*sizeof(int) );
	hipMemcpy(devTresholded,image->getArray(),size*sizeof(int),hipMemcpyHostToDevice);
	thresholdImage<<<blocks,threads>>>(devTresholded,image->getScale()/2,size);
	makeLaplaceMask<<<blocks,threads>>>(devTresholded, 
		devImage, size, image->getWidth(),image->getHeight());
	int n = 1000;
	float *ro;
	float *theta;
	float roStepSize = sqrt(image->getWidth()+image->getHeight())/n;
	float thetaStepSize = 3.14/n;
	hipMalloc((void**)&ro,sizeof(float)* n);
	hipMalloc((void**)&theta,sizeof(float)* n);
	createRoAndThetaArrays<<<blocks,threads>>>(ro, theta, 
		roStepSize, thetaStepSize, n);
	int *imageTres = new int[size];
	hipMemcpy(imageTres,devImage,size*sizeof(int), hipMemcpyDeviceToHost);
	for(int i=0;i<size;i++)
	{
		//printf("%d ",imageTres[i]);
	}
	int *A;
	hipMalloc((void**)&A,sizeof(int)*size );
	houghTransform<<<blocks,threads>>>(imageTres, ro, theta, 
		A, n, n, image->getWidth(),image->getHeight());
	int *index;
	hipMalloc((void**)&index,sizeof(int)*size );
	findLocalMaximas<<<blocks,threads>>>(A, 3, index, size);
	int *result = new int[size];
	hipMemcpy(result,index,size*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i=0;i<size;i++)
	{
		//printf("%d ",result[i]);
	}
	image->setArray(result);
	//saveImageToFile(result,resultFile);
	/*
	int n=1000;
=======

	Image *image=readImageFromFile("mountain.pgm");
	for(int i=0;i<10;i++)
	{
		for(int j=0;j<10;j++)
		{
			image->getArray()[i];
		}
	}
	/*int n=1000;
>>>>>>> ec5918de9505894795af882336c67e797ff38e0d
	char inputFile[] = { "mountain.pgm" };
	char outputFile[] = { "treshold.pgm" };
	char resultFile[] = { "Houghed.pgm" };
	Image *image=readImageFromFile(inputFile);
<<<<<<< HEAD
	/*char testFile[] = { "test.pgm" };
=======
	char testFile[] = { "test.pgm" };
>>>>>>> ec5918de9505894795af882336c67e797ff38e0d
	saveImageToFileTest(testFile);
	Image *devImage;
	Image *devThresholdImage;
	int blocks = image->getWidth();
	int threads = image->getHeight(); // height of the image is max 480
	Image *result = new Image(image->getWidth(),image->getHeight(),image->getScale());
	/*
	double *ro;
	double *theta;
	size_t steps = n*sizeof(int);
	int *A;
	int size=image->getHeight()*image->getWidth();
	size_t size2 = size*sizeof(int);
	int *indexes;
	int *indexesHost = new int[size];
	int *deviceImage;
	hipMalloc((void**)&devImage,sizeof(Image) );
	hipMalloc((void**)&deviceImage,size2 );
	hipMalloc((void**)&indexes,size2);
	hipMalloc((void**)&devThresholdImage,sizeof(Image) );
	hipMalloc((void**)&A,size2 );
	hipMalloc((void**)&ro,sizeof(double)* steps);
	hipMalloc((void**)&theta,sizeof(double)* steps);
	hipMemcpy(devImage,image,sizeof(Image),hipMemcpyHostToDevice);
	hipMemcpy(devThresholdImage,result,sizeof(Image),hipMemcpyHostToDevice);
	printf("End of allocation\n");
<<<<<<< HEAD
	//makeLaplaceMask<<<blocks,threads>>>(devImage,
	// devThresholdImage, size);
	thresholdImage<<<blocks,threads>>>(deviceImage,devThresholdImage->getScale()/2,size);
	int *imageTres = new int[size];
	hipMemcpy(imageTres,devThresholdImage,sizeof(Image), hipMemcpyDeviceToHost);
=======
	makeLaplaceMask<<<blocks,threads>>>(devImage,devThresholdImage, size);
	thresholdImage<<<blocks,threads>>>(devThresholdImage,devThresholdImage->getScale()/2,size);
	hipMemcpy(result,devThresholdImage,sizeof(Image), hipMemcpyDeviceToHost);
	hipMemcpy(B,deviceImage,size2, hipMemcpyDeviceToHost);
>>>>>>> ec5918de9505894795af882336c67e797ff38e0d
	for(int i=0;i<size;i++)
	{
		printf("%d ",imageTres[i]);
	}
	/*hipMemcpy(B,deviceImage,size2, hipMemcpyDeviceToHost);
	for(int i=0;i<size;i++)
	{
		printf("%i ",B[i]);
	}houghTransform<<<blocks,threads>>>(devThresholdImage,
	 ro, theta, A, steps, steps);
	printf("After treshold\n");
    saveImageToFile(result,outputFile);
	double roMax = sqrt(image->getHeight()*image->getHeight() + 
	image->getWidth()*image->getWidth());
	createRoAndThetaArrays<<<blocks,threads>>>(ro, theta, roMax/steps, 3.14/steps, steps);
	printf("After Ro theta arrays\n");
	houghTransform<<<blocks,threads>>>(devThresholdImage,
	 ro, theta, A, steps, steps);
	printf("Hough\n");
	findLocalMaximas<<<blocks,threads>>>(A, 5, indexes, size);
	printf("Local Maximas\n");
	//hipMemcpy(result,devImage,sizeof(Image), hipMemcpyDeviceToHost);
	hipMemcpy(indexesHost,indexes,sizeof(int)
		*image->getHeight()*image->getWidth(),
		hipMemcpyDeviceToHost);
	result->setArray(indexesHost);
	printf("After set array\n");
	saveImageToFile(result,resultFile);
	hipFree(theta);
	hipFree(ro);
	hipFree(A);
	hipFree(devImage);
	hipFree(deviceImage);
	hipFree(indexes);
	hipFree(devThresholdImage);
	free(image);
	free(result);
<<<<<<< HEAD
	//free(B);*/
=======
	free(B);*/
>>>>>>> ec5918de9505894795af882336c67e797ff38e0d
	return 0;
}
