#include "file.h"
#include "hough.h"


int main(int argc, char ** argv)
{

	Image *image=readImageFromFile("image.pgm");
	Image *devImage;
	Image *devThresholdImage;
	hipMalloc((void**)&devImage,sizeof(Image) );
	hipMalloc((void**)&devThresholdImage,sizeof(Image) );
	hipMemcpy(devImage,image,sizeof(Image),hipMemcpyHostToDevice);
	hipMemcpy(devThresholdImage,image,sizeof(Image),hipMemcpyHostToDevice);
	thresholdImage<<<10,10>>>(devImage,devThresholdImage,10);
/*
	int *deviceImage;
	int *deviceThresholdedImage;
        int *thresholdedImage;
	int threshold=10;
	int *deviceThreshold;
	int size=n*m;
	int *deviceSize;
	hipMalloc((void**)&deviceImage,
		size*sizeof(int) );
	hipMalloc((void**)&deviceThresholdedImage,
		size*sizeof(int) );
	hipMalloc((void**)&deviceThreshold,sizeof(int) );
	hipMalloc((void**)&deviceSize,sizeof(int) );
	hipMemcpy(deviceImage,
		image,size*sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(deviceSize,
		(const void*)&size,sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(deviceThreshold,
                (const void*)&threshold,sizeof(int),
                hipMemcpyHostToDevice);
	thresholdImage<<n,m>>(deviceImage,
		deviceThresholdedImage,
		deviceThreshold,deviceSize);
	hipMemcpy(thresholdedImage,
		deviceThresholdedImage,
		size*sizeof(int),
		hipMemcpyDeviceToHost);
	int i;
	for(i=0;i<size;i++)
	{
		printf("%d ",thresholdedImage[i]);
	}	*/
	return 0;
}
