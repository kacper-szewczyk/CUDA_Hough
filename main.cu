#include "file.h"
#include "hough.h"


int main(int argc, char ** argv)
{

	Image *image=readImageFromFile("image.pgm");
	Image *devImage;
	Image *devThresholdImage;
	Image *result = new Image();
	double *ro;
	double *theta;
	double steps = 1000;
	int *A;
	int *indexes;
	int *indexesHost;
	int size=image->getHeight()*image->getWidth();
	hipMalloc((void**)&devImage,sizeof(Image) );
	hipMalloc((void**)&indexes,sizeof(int)*size);
	hipMalloc((void**)&devThresholdImage,sizeof(Image) );
	hipMalloc((void**)&A,sizeof(int) );
	hipMalloc((void**)&ro,sizeof(double)* steps);
	hipMalloc((void**)&theta,sizeof(double)* steps);
	hipMemcpy(devImage,image,sizeof(Image),hipMemcpyHostToDevice);
	hipMemcpy(devThresholdImage,image,sizeof(Image),hipMemcpyHostToDevice);
	thresholdImage<<<10,10>>>(devImage,devThresholdImage,10);
	createRoAndThetaArrays<<<10,10>>>(ro, theta, 3.14/steps, 3.14/steps, steps);
	houghTransform<<<10,10>>>(devThresholdImage,
	 ro, theta, A, 100, 100);
	findLocalMaximas<<<10,10>>>(A, 5, indexes);
	hipMemcpy(result,devThresholdImage,sizeof(Image),hipMemcpyDeviceToHost);
	hipMemcpy(indexesHost,indexes,sizeof(int)
		*image->getHeight()*image->getWidth(),
		hipMemcpyDeviceToHost);
	result->setArray(indexesHost);
	saveImageToFile(result,"Houghed.pgm");
	//houghTransform<<<10,10>>>(devThresholdImage, ro, theta, int *A, int R, int T)
/*
	int *deviceImage;
	int *deviceThresholdedImage;
        int *thresholdedImage;
	int threshold=10;
	int *deviceThreshold;
	int size=n*m;
	int *deviceSize;
	hipMalloc((void**)&deviceImage,
		size*sizeof(int) );
	hipMalloc((void**)&deviceThresholdedImage,
		size*sizeof(int) );
	hipMalloc((void**)&deviceThreshold,sizeof(int) );
	hipMalloc((void**)&deviceSize,sizeof(int) );
	hipMemcpy(deviceImage,
		image,size*sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(deviceSize,
		(const void*)&size,sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(deviceThreshold,
                (const void*)&threshold,sizeof(int),
                hipMemcpyHostToDevice);
	thresholdImage<<n,m>>(deviceImage,
		deviceThresholdedImage,
		deviceThreshold,deviceSize);
	hipMemcpy(thresholdedImage,
		deviceThresholdedImage,
		size*sizeof(int),
		hipMemcpyDeviceToHost);
	int i;
	for(i=0;i<size;i++)
	{
		printf("%d ",thresholdedImage[i]);
	}	*/
	return 0;
}
