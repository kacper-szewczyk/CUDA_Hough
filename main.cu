#include "hip/hip_runtime.h"
#include "file.h"
#include "hough.h"
#include "laplace.h"

int main(int argc, char ** argv)
{
	char inputFile[] = { "mountain.pgm" };
	char resultFile[] = { "Houghed.pgm" };
	Image *image=readImageFromFile(inputFile);
	int *devImage;
	int *devTresholded;
	int blocks = image->getWidth();
	int threads = image->getHeight();
	int size=image->getHeight()*image->getWidth();
	//size_t imageSize = (size+3) * sizeof(int);
	hipMalloc((void**)&devImage,size*sizeof(int) );
	hipMemcpy(devImage,image->getArray(),size*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc((void**)&devTresholded,size*sizeof(int) );
	hipMemcpy(devTresholded,image->getArray(),size*sizeof(int),hipMemcpyHostToDevice);
	thresholdImage<<<blocks,threads>>>(devTresholded,image->getScale()/2,size);
	makeLaplaceMask<<<blocks,threads>>>(devTresholded, 
		devImage, size, image->getWidth(),image->getHeight());
	int n = 1000;
	float *ro;
	float *theta;
	float roStepSize = sqrt(image->getWidth()+image->getHeight())/n;
	float thetaStepSize = 3.14/n;
	hipMalloc((void**)&ro,sizeof(float)* n);
	hipMalloc((void**)&theta,sizeof(float)* n);
	createRoAndThetaArrays<<<blocks,threads>>>(ro, theta, 
		roStepSize, thetaStepSize, n);
	int *imageTres = new int[size];
	hipMemcpy(imageTres,devImage,size*sizeof(int), hipMemcpyDeviceToHost);
	for(int i=0;i<size;i++)
	{
		//printf("%d ",imageTres[i]);
	}
	int *A;
	hipMalloc((void**)&A,sizeof(int)*size );
	houghTransform<<<blocks,threads>>>(imageTres, ro, theta, 
		A, n, n, image->getWidth(),image->getHeight());
	int *index;
	hipMalloc((void**)&index,sizeof(int)*size );
	findLocalMaximas<<<blocks,threads>>>(A, 3, index, size);
	int *result = new int[size];
	hipMemcpy(result,index,size*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i=0;i<size;i++)
	{
		//printf("%d ",result[i]);
	}
	image->setArray(result);
	saveImageToFile(image,resultFile);
	
	return 0;
}
