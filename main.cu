#include "hip/hip_runtime.h"
#include "file.h"
#include "hough.h"
#include "laplace.h"

int main(int argc, char ** argv)
{

	Image *image=readImageFromFile("mountain.pgm");
	for(int i=0;i<10;i++)
	{
		for(int j=0;j<10;j++)
		{
			image->getArray()[i];
		}
	}
	/*int n=1000;
	char inputFile[] = { "mountain.pgm" };
	char outputFile[] = { "treshold.pgm" };
	char resultFile[] = { "Houghed.pgm" };
	Image *image=readImageFromFile(inputFile);
	char testFile[] = { "test.pgm" };
	saveImageToFileTest(testFile);
	Image *devImage;
	Image *devThresholdImage;
	Image *result = new Image(image->getWidth(),image->getHeight(),image->getScale());
	int blocks = image->getWidth();
	int threads = image->getHeight(); // height of the image is max 480
	double *ro;
	double *theta;
	size_t steps = n*sizeof(int);
	int *A;
	int size=image->getHeight()*image->getWidth();
	size_t size2 = size*sizeof(int);
	int *indexes;
	int *indexesHost = new int[size];
	int *deviceImage;
	int *B = new int[size];
	hipMalloc((void**)&devImage,sizeof(Image) );
	hipMalloc((void**)&deviceImage,size2 );
	hipMalloc((void**)&indexes,size2);
	hipMalloc((void**)&devThresholdImage,sizeof(Image) );
	hipMalloc((void**)&A,size2 );
	hipMalloc((void**)&ro,sizeof(double)* steps);
	hipMalloc((void**)&theta,sizeof(double)* steps);
	hipMemcpy(devImage,image,sizeof(Image),hipMemcpyHostToDevice);
	hipMemcpy(devThresholdImage,result,sizeof(Image),hipMemcpyHostToDevice);
	printf("End of allocation\n");
	makeLaplaceMask<<<blocks,threads>>>(devImage,devThresholdImage, size);
	thresholdImage<<<blocks,threads>>>(devThresholdImage,devThresholdImage->getScale()/2,size);
	hipMemcpy(result,devThresholdImage,sizeof(Image), hipMemcpyDeviceToHost);
	hipMemcpy(B,deviceImage,size2, hipMemcpyDeviceToHost);
	for(int i=0;i<size;i++)
	{
		printf("%i ",B[i]);
	}
	printf("After treshold\n");
    saveImageToFile(result,outputFile);
	double roMax = sqrt(image->getHeight()*image->getHeight() + 
	image->getWidth()*image->getWidth());
	createRoAndThetaArrays<<<blocks,threads>>>(ro, theta, roMax/steps, 3.14/steps, steps);
	printf("After Ro theta arrays\n");
	houghTransform<<<blocks,threads>>>(devThresholdImage,
	 ro, theta, A, steps, steps);
	printf("Hough\n");
	findLocalMaximas<<<blocks,threads>>>(A, 5, indexes, size);
	printf("Local Maximas\n");
	//hipMemcpy(result,devImage,sizeof(Image), hipMemcpyDeviceToHost);
	hipMemcpy(indexesHost,indexes,sizeof(int)
		*image->getHeight()*image->getWidth(),
		hipMemcpyDeviceToHost);
	result->setArray(indexesHost);
	printf("After set array\n");
	saveImageToFile(result,resultFile);
	hipFree(theta);
	hipFree(ro);
	hipFree(A);
	hipFree(devImage);
	hipFree(deviceImage);
	hipFree(indexes);
	hipFree(devThresholdImage);
	free(image);
	free(result);
	free(B);*/
	return 0;
}
