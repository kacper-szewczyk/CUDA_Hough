#include "file.h"
#include "hough.h"


int main(int argc, char ** argv)
{

	Image *image=readImageFromFile("image.pgm");
	Image *devImage;
	Image *devThresholdImage;
	double *ro;
	double *theta;
	double steps = 1000;
	int *A;
	hipMalloc((void**)&devImage,sizeof(Image) );
	hipMalloc((void**)&devThresholdImage,sizeof(Image) );
	hipMalloc((void**)&A,sizeof(int) );
	hipMalloc((void**)&ro,sizeof(double)* steps);
	hipMalloc((void**)&theta,sizeof(double)* steps);
	hipMemcpy(devImage,image,sizeof(Image),hipMemcpyHostToDevice);
	hipMemcpy(devThresholdImage,image,sizeof(Image),hipMemcpyHostToDevice);
	thresholdImage<<<10,10>>>(devImage,devThresholdImage,10);
	createRoAndThetaArrays<<<10,10>>>(ro, theta, 3.14/steps, 3.14/steps, steps);
	//houghTransform<<<10,10>>>(devThresholdImage, ro, theta, int *A, int R, int T)
/*
	int *deviceImage;
	int *deviceThresholdedImage;
        int *thresholdedImage;
	int threshold=10;
	int *deviceThreshold;
	int size=n*m;
	int *deviceSize;
	hipMalloc((void**)&deviceImage,
		size*sizeof(int) );
	hipMalloc((void**)&deviceThresholdedImage,
		size*sizeof(int) );
	hipMalloc((void**)&deviceThreshold,sizeof(int) );
	hipMalloc((void**)&deviceSize,sizeof(int) );
	hipMemcpy(deviceImage,
		image,size*sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(deviceSize,
		(const void*)&size,sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(deviceThreshold,
                (const void*)&threshold,sizeof(int),
                hipMemcpyHostToDevice);
	thresholdImage<<n,m>>(deviceImage,
		deviceThresholdedImage,
		deviceThreshold,deviceSize);
	hipMemcpy(thresholdedImage,
		deviceThresholdedImage,
		size*sizeof(int),
		hipMemcpyDeviceToHost);
	int i;
	for(i=0;i<size;i++)
	{
		printf("%d ",thresholdedImage[i]);
	}	*/
	return 0;
}
