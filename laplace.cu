#include "hip/hip_runtime.h"
/*
 * laplace.cu
 *
 *  Created on: 28 maj 2015
 *      Author: kszewcz2
 */

#include "laplace.h"

__device__ double laplaceMaskOnSector(Image *originalImage, int index, int width, int height)
{
	double sum = 0;
	if(index>0)
	{
		sum += (double)4*originalImage->array[index-1];
	}
	if(index<width*height)
	{
		sum += (double)4*originalImage->array[index+1];
	}
	if(index>=width)
	{
		sum += (double)4*originalImage->array[index-width];
		sum += (double)1*originalImage->array[index-width+1];
	}
	if(index>=width+1)
	{
		sum += (double)1*originalImage->array[index-width-1];
	}
	if(index<=width*(height-1))
	{
		sum += (double)4*originalImage->array[index+width];
		sum += (double)1*originalImage->array[index+width-1];
	}
	if(index<=width*(height-1)-1)
	{
		sum += (double)1*originalImage->array[index+width+1];
	}
	return sum/6;
}

__global__ void makeLaplaceMask(Image *originalImage,
	 Image *image, int n)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;
	int width = originalImage->width;
	int height = originalImage->height;
	for(int i=offset;i<n;i+=blockIdx.x * blockDim.x)
	{
		
		image->array[i] = (int)laplaceMaskOnSector(originalImage, i, width, height);
	}
	//__syncthreads();
}
