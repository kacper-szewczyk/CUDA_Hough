#include "hip/hip_runtime.h"
/*
 * laplace.cu
 *
 *  Created on: 28 maj 2015
 *      Author: kszewcz2
 */

#include "laplace.h"

__device__ int laplaceMaskOnSector(int *originalImage, int index, int width, int height)
{
	int sum = 0;
	if(index>0)
	{
		sum += (double)4*originalImage[index-1];
	}
	if(index<width*height)
	{
		sum += (double)4*originalImage[index+1];
	}
	if(index>=width)
	{
		sum += (double)4*originalImage[index-width];
		sum += (double)1*originalImage[index-width+1];
	}
	if(index>=width+1)
	{
		sum += (double)1*originalImage[index-width-1];
	}
	if(index<=width*(height-1))
	{
		sum += (double)4*originalImage[index+width];
		sum += (double)1*originalImage[index+width-1];
	}
	if(index<=width*(height-1)-1)
	{
		sum += (double)1*originalImage[index+width+1];
	}
	return sum/6;
}

__global__ void makeLaplaceMask(int *originalImage,
	 int *image, int n, int width, int height)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;

	if( offset <n )
	{
		image[offset] = laplaceMaskOnSector(originalImage, offset, width, height);
		offset = threadIdx.x + blockIdx.x * blockDim.x;
	}
	//__syncthreads();
}
