#include "hip/hip_runtime.h"
/*
 * hough.c
 *
 *  Created on: 5 maj 2015
 *      Author: kszewcz2
 */
#include "hough.h"

__global__ void thresholdImage(Image *deviceImage,Image *deviceThresholdedImage, int threshold)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	int * array=deviceImage->getArray();
	for(int i=offset;i<deviceImage->getWidth()*deviceImage->getHeight();i+= blockDim.x * gridDim.x){
		
		if(array[i]>=threshold)
			array[i] = 1;
		else
			array[i] = 0;
		offset += blockDim.x * gridDim.x;

	}
	
}
