#include "hip/hip_runtime.h"
/*
 * hough.c
 *
 *  Created on: 5 maj 2015
 *      Author: kszewcz2
 */
#include "hough.h"

__global__ void thresholdImage(int *deviceImage,
		int *deviceThresholdedImage, int *threshold, int *size)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	while(offset < (*size))
	{
		if(deviceImage[offset]>=(*threshold))
			deviceThresholdedImage[offset] = 1;
		else
			deviceThresholdedImage[offset] = 0;
		offset += blockDim.x * gridDim.x;
	}
}
