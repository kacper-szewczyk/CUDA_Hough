#include "hip/hip_runtime.h"
/*
 * hough.cu
 *
 *  Created on: 5 maj 2015
 *      Author: kszewcz2
 */
#include "hough.h"
#include <stdio.h>
__global__ void thresholdImage(int *deviceThresholdedImage, int threshold, int n)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;	
	/*	
	for(int i=offset;i<n; i+= blockDim.x * gridDim.x)
	{
		if(deviceThresholdedImage->array[i]>=threshold)
			deviceThresholdedImage->array[i] = 1;
		else
			deviceThresholdedImage->array[i] = 0;
		offset += blockDim.x * gridDim.x;
	}*/
	if(offset < n)	
	//for(int i=offset;i<n; i+= blockDim.x * gridDim.x)
	{
		if(deviceThresholdedImage[offset]>=threshold)
			deviceThresholdedImage[offset] = 1;
		else
			deviceThresholdedImage[offset] = 0;
		offset += blockDim.x * gridDim.x;
	}
	//deviceThresholdedImage->array[offset] = 5;
}

__global__ void createRoAndThetaArrays(float *ro, float *theta, float roStepSize, float thetaStepSize, int steps)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;
	if( offset < steps )	
	{
		ro[offset] = roStepSize * offset;
		theta[offset] = thetaStepSize * offset;
		offset += blockDim.x * gridDim.x;
	}
}

__device__ int findMaxWidth(int i, int width)
{
	int result = 0;
	int summed = 0;
	while(summed <= i)
	{
		summed += width;
		result++;
	}
	if(summed > i)
	{
		result--;
	}
	return result;
}

__global__ void houghTransform(int *array,
	 float *ro, float *theta, int *A, int R, int T, int width, int height)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;
	float roIdeal;
	float roCandidate;
	int kRoClosest;	
	float difference=9999;
	int indexI,indexJ;
	if(offset < width*height)
	{
		indexI = findMaxWidth(offset,width);
		indexJ = offset-indexI*width;
		if(array[offset] == 1)
		{
			for(int h=0;h<T;h++)
			{
				roIdeal = indexI*sin(theta[h])+indexJ*cos(theta[h]);
				for(int k=0;k<R;k++)
				{
					roCandidate = abs(roIdeal - ro[k]); 
					if(roCandidate<difference)
					{
						difference = roCandidate;
						kRoClosest = k;
					}
				}
				A[kRoClosest*R+h]++;
				difference=9999;
			}
		}
 
		offset += blockDim.x * gridDim.x;

	}
}

__global__ void findLocalMaximas(int *A, int threshold, int *indexes, int size)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;
	if(offset < size)
	{
		if(A[offset]>=threshold)
		{
			printf("%d",A[offset]);
			indexes[offset]=1;
		}
		else
		{
			indexes[offset]=0;
		}
		offset += blockDim.x * gridDim.x;
	}
}
