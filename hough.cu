#include "hip/hip_runtime.h"
/*
 * hough.cu
 *
 *  Created on: 5 maj 2015
 *      Author: kszewcz2
 */
#include "hough.h"

__global__ void thresholdImage(Image *deviceThresholdedImage, int threshold, int n)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;	
	/*	
	for(int i=offset;i<n; i+= blockDim.x * gridDim.x)
	{
		if(deviceThresholdedImage->array[i]>=threshold)
			deviceThresholdedImage->array[i] = 1;
		else
			deviceThresholdedImage->array[i] = 0;
		offset += blockDim.x * gridDim.x;
	}*/
	while(offset < n)	
	//for(int i=offset;i<n; i+= blockDim.x * gridDim.x)
	{
		if(deviceThresholdedImage->array[offset]>=threshold)
			deviceThresholdedImage->array[offset] = 1;
		else
			deviceThresholdedImage->array[offset] = 0;
		offset += blockDim.x * gridDim.x;
	}
	//deviceThresholdedImage->array[offset] = 5;
}

__global__ void createRoAndThetaArrays(double *ro, double *theta, double roStepSize, double thetaStepSize, double steps)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;
	/*
	for(int i=offset; i<=steps; i+= blockDim.x * gridDim.x)
	{
		ro[i] = roStepSize * i;
		theta[i] = thetaStepSize * i;
		offset += blockDim.x * gridDim.x;
	}*/
	while(offset < steps)	
	{
		ro[offset] = roStepSize * offset;
		theta[offset] = thetaStepSize * offset;
		offset += blockDim.x * gridDim.x;
	}
}

__device__ int findMaxWidth(int i, int width)
{
	int result = 0;
	int summed = 0;
	while(summed <= i)
	{
		summed += width;
		result++;
	}
	if(summed > i)
	{
		result--;
	}
	return result;
}

__global__ void houghTransform(Image *deviceThresholdedImage,
	 double *ro, double *theta, int *A, int R, int T)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;
	int * array=deviceThresholdedImage->array;
	double roIdeal;
	double roCandidate;
	int kRoClosest;	
	double difference=9999;
	int indexI,indexJ;
	int width = deviceThresholdedImage->width;
	while(offset < width*deviceThresholdedImage->height)
	{
		indexI = findMaxWidth(offset,width);
		indexJ = offset-indexI*width;
		if(array[offset] == 1)
		{
			for(int h=0;h<T;h++)
			{
				roIdeal = indexI*sin(theta[h])+indexJ*cos(theta[h]);
				for(int k=0;k<R;k++)
				{
					roCandidate = abs(roIdeal - ro[k]); 
					if(roCandidate<difference)
					{
						difference = roCandidate;
						kRoClosest = k;
					}
				}
				A[kRoClosest*R+h]++;
				difference=9999;
			}
		}
 
		offset += blockDim.x * gridDim.x;

	}
}

__global__ void findLocalMaximas(int *A, int threshold, int *indexes, int size)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;
	while(offset < size)
	{
		if(A[offset]>threshold)
		{
			indexes[offset]=1;
		}
		offset += blockDim.x * gridDim.x;
	}
}
